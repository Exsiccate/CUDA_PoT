﻿#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <iostream>
#include <time.h>

int main(void)
{
    // allocate three device_vectors with 10 elements
    thrust::device_vector<__int32> PoT(300001);
    thrust::device_vector<__int32> Rest(300001);
    thrust::device_vector<__int32> Twos(300001);
    thrust::device_vector<__int32> Divs(300001);
    thrust::device_vector<__int32> Mults(300001);

    // fill last element with 1 
    PoT[300000] = 1;

    // fill Twos with twos
    thrust::fill(Twos.begin(), Twos.end(), 2);
    // fill Divs with twos
    thrust::fill(Divs.begin(), Divs.end(), 1000000000);

    struct tm newtime;
    __time64_t long_time;
    char timebuf[26];

    // Get time as 64-bit integer.
    _time64(&long_time);
    // Convert to local time.
    _localtime64_s(&newtime, &long_time);
    asctime_s(timebuf, 26, &newtime);
    printf("%.19s \n", timebuf);

    for (int i = 0; i < 10000; i++)
    {
        // compute PoT = PoT * 2 [in form of vector named twos]
        thrust::transform(PoT.begin(), PoT.end(), Twos.begin(), PoT.begin(), thrust::multiplies<__int32>());
        thrust::transform(PoT.begin(), PoT.end(), Divs.begin(), Rest.begin(), thrust::divides<__int32>());
        thrust::transform(Rest.begin(), Rest.end(), Divs.begin(), Mults.begin(), thrust::multiplies<__int32>());
        thrust::transform(PoT.begin(), PoT.end(), Mults.begin(), PoT.begin(), thrust::minus<__int32>());
        thrust::transform(PoT.begin(), PoT.end(), Rest.begin() + 1, PoT.begin(), thrust::plus<__int32>());
        // print PoT
        //thrust::copy(PoT.begin(), PoT.end(), std::ostream_iterator<__int32>(std::cout, ","));
        // print Rest
        //std::cout << i;
        // print \n
        //std::cout << "\n";
    }

    // Get time as 64-bit integer.
    _time64(&long_time);
    // Convert to local time.
    _localtime64_s(&newtime, &long_time);
    asctime_s(timebuf, 26, &newtime);
    printf("%.19s \n", timebuf);

    return 0;
}